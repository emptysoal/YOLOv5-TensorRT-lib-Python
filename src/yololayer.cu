#include "hip/hip_runtime.h"
﻿#include <assert.h>
#include "yololayer.h"

namespace Tn
{
    template<typename T> 
    void write(char*& buffer, const T& val)
    {
        *reinterpret_cast<T*>(buffer) = val;
        buffer += sizeof(T);
    }

    template<typename T> 
    void read(const char*& buffer, T& val)
    {
        val = *reinterpret_cast<const T*>(buffer);
        buffer += sizeof(T);
    }
}

using namespace Yolo;

namespace nvinfer1
{
    YoloLayerPlugin::YoloLayerPlugin(int classCount, int netWidth, int netHeight, int maxOut, const std::vector<Yolo::YoloKernel>& vYoloKernel)
    {
        mClassCount = classCount;
        mYoloV5NetWidth = netWidth;
        mYoloV5NetHeight = netHeight;
        mMaxOutObject = maxOut;
        mYoloKernel = vYoloKernel;
        mKernelCount = vYoloKernel.size();

        CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float) * CHECK_COUNT * 2;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
    }

    YoloLayerPlugin::~YoloLayerPlugin()
    {
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CHECK(hipFree(mAnchor[ii]));
        }
        CHECK(hipHostFree(mAnchor));
    }

    // create the plugin at runtime from a byte stream
    YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length)
    {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        read(d, mYoloV5NetWidth);
        read(d, mYoloV5NetHeight);
        read(d, mMaxOutObject);
        mYoloKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(mYoloKernel.data(), d, kernelSize);
        d += kernelSize;
        CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* CHECK_COUNT * 2;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
        assert(d == a + length);
    }

    void YoloLayerPlugin::serialize(void *buffer) const noexcept
    {
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        write(d, mYoloV5NetWidth);
        write(d, mYoloV5NetHeight);
        write(d, mMaxOutObject);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(d, mYoloKernel.data(), kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }

    size_t YoloLayerPlugin::getSerializationSize() const noexcept
    {
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount) + sizeof(Yolo::YoloKernel) * mYoloKernel.size() + sizeof(mYoloV5NetWidth) + sizeof(mYoloV5NetHeight) + sizeof(mMaxOutObject);
    }

    int32_t YoloLayerPlugin::initialize() noexcept
    {
        return 0;
    }

    DimsExprs YoloLayerPlugin::getOutputDimensions(int32_t outputIndex, const DimsExprs* inputs, int32_t nbInputs, IExprBuilder &exprBuilder) noexcept
    {
        //output the result to channel
        int totalsize = mMaxOutObject * sizeof(Detection) / sizeof(float);
        // return Dims32{4, {1, totalsize + 1, 1, 1}};
        return DimsExprs{4, {exprBuilder.constant(1), exprBuilder.constant(totalsize + 1), exprBuilder.constant(1), exprBuilder.constant(1)}};
    }

    void YoloLayerPlugin::setPluginNamespace(const char* pluginNamespace) noexcept
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* YoloLayerPlugin::getPluginNamespace() const noexcept
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType YoloLayerPlugin::getOutputDataType(int32_t index, DataType const *inputTypes, int32_t nbInputs) const noexcept
    {
        return DataType::kFLOAT;
    }

    void YoloLayerPlugin::configurePlugin(const DynamicPluginTensorDesc *in, int32_t nbInputs, const DynamicPluginTensorDesc *out, int32_t nbOutputs) noexcept
    {
    }

    void YoloLayerPlugin::attachToContext(cudnnContext *contextCudnn, cublasContext *contextCublas, IGpuAllocator *gpuAllocator) noexcept
    {
    }

    void YoloLayerPlugin::detachFromContext() noexcept
    {
    }

    const char* YoloLayerPlugin::getPluginType() const noexcept
    {
        return "YoloLayer_TRT";
    }

    const char* YoloLayerPlugin::getPluginVersion() const noexcept
    {
        return "1";
    }

    void YoloLayerPlugin::destroy() noexcept
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2DynamicExt* YoloLayerPlugin::clone() const noexcept
    {
        YoloLayerPlugin* p = new YoloLayerPlugin(mClassCount, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, mYoloKernel);
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float Logist(float data)
    {
        return 1.0f / (1.0f + expf(-data));
    }

    __global__ void CalDetection(const float* input, float* output, int noElements,
        const int netWidth, const int netHeight, int maxoutobject, int yoloWidth, int yoloHeight, const float anchors[CHECK_COUNT * 2], int classes, int outputElem)
    {
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;

        int total_grid = yoloWidth * yoloHeight;
        int bnIdx = idx / total_grid;
        idx = idx - total_grid * bnIdx;
        int info_len_i = 5 + classes;
        const float* curInput = input + bnIdx * (info_len_i * total_grid * CHECK_COUNT);

        for (int k = 0; k < CHECK_COUNT; k++)
        {
            float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
            if (box_prob < IGNORE_THRESH) continue;
            int class_id = 0;
            float max_cls_prob = 0.0;
            for (int i = 5; i < info_len_i; i++)
            {
                float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
                if (p > max_cls_prob)
                {
                    max_cls_prob = p;
                    class_id = i - 5;
                }
            }
            float* res_count = output + bnIdx * outputElem;
            int count = (int)atomicAdd(res_count, 1);
            if (count >= maxoutobject) return;
            char* data = (char*)res_count + sizeof(float) + count * sizeof(Detection);
            Detection* det = (Detection*)(data);

            int row = idx / yoloWidth;
            int col = idx % yoloWidth;

            //Location
            // pytorch:
            //  y = x[i].sigmoid()
            //  y[..., 0:2] = (y[..., 0:2] * 2. - 0.5 + self.grid[i].to(x[i].device)) * self.stride[i]  # xy
            //  y[..., 2:4] = (y[..., 2:4] * 2) ** 2 * self.anchor_grid[i]  # wh
            //  X: (sigmoid(tx) + cx)/FeaturemapW *  netwidth
            det->bbox[0] = (col - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * netWidth / yoloWidth;
            det->bbox[1] = (row - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * netHeight / yoloHeight;

            // W: (Pw * e^tw) / FeaturemapW * netwidth
            // v5: https://github.com/ultralytics/yolov5/issues/471
            det->bbox[2] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]);
            det->bbox[2] = det->bbox[2] * det->bbox[2] * anchors[2 * k];
            det->bbox[3] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]);
            det->bbox[3] = det->bbox[3] * det->bbox[3] * anchors[2 * k + 1];
            det->conf = box_prob * max_cls_prob;
            det->class_id = class_id;
        }
    }

    void YoloLayerPlugin::forwardGpu(const float* const* inputs, float *output, hipStream_t stream, int batchSize)
    {
        int outputElem = 1 + mMaxOutObject * sizeof(Detection) / sizeof(float);
        for (int idx = 0; idx < batchSize; ++idx) {
            CHECK(hipMemsetAsync(output + idx * outputElem, 0, sizeof(float), stream));
        }
        int numElem = 0;
        for (unsigned int i = 0; i < mYoloKernel.size(); ++i) {
            const auto& yolo = mYoloKernel[i];
            numElem = yolo.width * yolo.height * batchSize;
            if (numElem < mThreadCount) mThreadCount = numElem;

            //printf("Net: %d  %d \n", mYoloV5NetWidth, mYoloV5NetHeight);
            CalDetection << < (numElem + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream >> >
                (inputs[i], output, numElem, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, yolo.width, yolo.height, (float*)mAnchor[i], mClassCount, outputElem);
        }
    }

    int32_t YoloLayerPlugin::enqueue(const PluginTensorDesc* inputDesc, const PluginTensorDesc* outputDesc, const void* const *inputs, void* const *outputs, void* workspace, hipStream_t stream) noexcept
    {
        forwardGpu((const float* const*)inputs, (float*)outputs[0], stream, 1);  // define batchsize = 1
        return 0;
    }

    PluginFieldCollection YoloPluginCreator::mFC{};
    std::vector<PluginField> YoloPluginCreator::mPluginAttributes;

    YoloPluginCreator::YoloPluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* YoloPluginCreator::getPluginName() const noexcept
    {
        return "YoloLayer_TRT";
    }

    const char* YoloPluginCreator::getPluginVersion() const noexcept
    {
        return "1";
    }

    const PluginFieldCollection* YoloPluginCreator::getFieldNames() noexcept
    {
        return &mFC;
    }

    IPluginV2DynamicExt* YoloPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) noexcept
    {
        assert(fc->nbFields == 2);
        assert(strcmp(fc->fields[0].name, "netinfo") == 0);
        assert(strcmp(fc->fields[1].name, "kernels") == 0);
        int *p_netinfo = (int*)(fc->fields[0].data);
        int class_count = p_netinfo[0];
        int input_w = p_netinfo[1];
        int input_h = p_netinfo[2];
        int max_output_object_count = p_netinfo[3];
        std::vector<Yolo::YoloKernel> kernels(fc->fields[1].length);
        memcpy(&kernels[0], fc->fields[1].data, kernels.size() * sizeof(Yolo::YoloKernel));
        YoloLayerPlugin* obj = new YoloLayerPlugin(class_count, input_w, input_h, max_output_object_count, kernels);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2DynamicExt* YoloPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength) noexcept
    {
        // This object will be deleted when the network is destroyed, which will
        // call YoloLayerPlugin::destroy()
        YoloLayerPlugin* obj = new YoloLayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

}  // namespace nvinfer1
